#include "hip/hip_runtime.h"
#include <thread>
#include <chrono>
#include <time.h>
#include <iostream>
#include <math.h>
#include <string>
#include <opencv2/opencv.hpp>
#include <opencv2/core/utility.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/imgcodecs.hpp>

using namespace cv;
using namespace std;

//Numero de hilos por bloque
#define N_THREADS 32.0

__global__ void filter_Sobel(unsigned char* src_img,unsigned char* out_img, unsigned int width, unsigned int height) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    float Gx, Gy; //Kernel para las direcciones x e y
    float G;
    float G_0_0,G_0_1,G_0_2,G_1_0,G_1_1,G_1_2,G_2_0,G_2_1,G_2_2;

    /* Comprobar los limites de la imagen */
    if(idx > 0 && idy > 0 && idx < width-1 && idy < height-1) { 

        /***********************************   Multiplicamos los valores de cada gradiente por la posición correspondiente (idx,idy) de la imagen original
              -1 0 +1              -1 -2 -1    Iteramos por los pixeles de la imagen mediante el uso de idx, idy y el ancho de la imagen
        Gx => -2 0 +2        Gy =>  0  0  0     
              -1 0 +1              +1 +2 +1
        ***********************************/
        G_0_0 = src_img[(idy-1)*width + (idx-1)];                
        G_0_1 = src_img[(idy-1)*width + (idx)];                       
        G_0_2 = src_img[(idy-1)*width + (idx+1)];                      
        G_1_0 = src_img[(idy)*width + (idx-1)];                        
        G_1_1 = src_img[(idy)*width + (idx)];                          
        G_1_2 = src_img[(idy)*width + (idx+1)];                        
        G_2_0 = src_img[(idy+1)*width + (idx-1)];                      
        G_2_1 = src_img[(idy+1)*width + (idx)];                        
        G_2_2 = src_img[(idy+1)*width + (idx+1)];                      

        Gx = (-1 * G_0_0) + (0 * G_0_1) + (1 * G_0_2) +
             (-2 * G_1_0) + (0 * G_1_1) + (2 * G_1_2) +
             (-1 * G_2_0) + (0 * G_2_1) + (1 * G_2_2);

        Gy = (1 * G_0_0) + (2 * G_0_1) + (1 * G_0_2) +
             (0 * G_1_0) + (0 * G_1_1) + (0 * G_1_2) +
             (-1 * G_2_0) + (-2 * G_2_1) + (-1 * G_2_2);
        
        /* El gradiente resultante (G) es la raiz cuadrada de (Gx^2 + Gy^2) */
        G = sqrt(pow(Gx,2) + pow(Gy,2));

        /* Modificamos el pixel que estamos comprobando */
        if(G > 255){  
            out_img[idy*width + idx] = 255;     //En caso de que sobrepasemos el valor maximo posible para este pixel (255), ponemos este ultimo como su valor actual
        }else{
            out_img[idy*width + idx] = G;       //Cualquier otro caso -> valor correspondiente
        }
    }
}

int main(int argc, char*argv[]) {
    /** Comprobar linea de comandos **/
    if(argc < 2 || argc > 3) {
        printf("\033[1;31mError: Invalid number of command line arguments.\nUsage: %s [image.png] [filter_option]\033[0m \n", argv[0]);
        return 1;
    }
    /** Propiedades de nuestro dispositvo CUDA **/
	hipDeviceProp_t dev_properties;
	hipGetDeviceProperties(&dev_properties, 0);
	int cores = dev_properties.multiProcessorCount;
	switch (dev_properties.major)
	{
	case 2: // Fermi
		if (dev_properties.minor == 1) cores *= 48;
		else cores *= 32; break;
	case 3: // Kepler
		cores *= 192; break;
	case 5: // Maxwell
		cores *= 128; break;
	case 6: // Pascal
		if (dev_properties.minor == 1) cores *= 128;
		else if (dev_properties.minor == 0) cores *= 64;
        break;
    case 7: // Volta and Turing
        if ((dev_properties.minor == 0) || (dev_properties.minor == 5)) cores *= 64;
        else printf("Unknown device type\n");
        break;
    case 8: // Ampere
        if (dev_properties.minor == 0) cores *= 64;
        else if (dev_properties.minor == 6) cores *= 128;
        else printf("Unknown device type\n");
        break;
    default:
        printf("Unknown device type\n"); 
        break;
    }
    
    /** Imprimir informacion (hardware threads, GPU info, etc) **/
    printf("CPU: %d hardware threads\n", std::thread::hardware_concurrency());
    printf("GPU: %s, CUDA %d.%d, %zd Mbytes global memory, %d CUDA cores\n",
    dev_properties.name, dev_properties.major, dev_properties.minor, dev_properties.totalGlobalMem / 1048576, cores);

    /** Buscar nuestra imagen **/
    string image_path = "./input_images/";

    /** Cargar nuestra imagen en escala de grises **/
    Mat original_img = imread(image_path + argv[1],IMREAD_GRAYSCALE);
    //Mat original_img = imread(image_path + argv[1], IMREAD_GRAYSCALE);
    if(original_img.empty()){
        printf("\033[1;31mError: Image not found\nPlease make sure it's on the \"input_images\" folder\033[0m \n");
        return 1;
    }

    /** En caso de que pasemos un filtro Gaussiano para suavizar el ruido de la imagen y mejorar su resultado **/
    Mat modified_img;
    if (argc == 3){
        int gauss_size = atoi(argv[2]);

        if(gauss_size % 2 == 0){
            printf("\033[1;31mError: Gauss filter size must be an odd number (Ej: 3,5,7,etc)\033[0m \n"); //Impar, ya que el kernel debe ser simetrico
            return 1;
        }
        GaussianBlur(original_img,modified_img,Size(gauss_size,gauss_size),0);

    /** Para la ejecucion normal del filtro Sobel **/
    }else{
        modified_img = original_img;
    }


    /** Datos que necesitamos de la imagen **/
    unsigned int img_data_height= modified_img.rows;
    unsigned int img_data_width = modified_img.cols;
    int img_data_size = img_data_width * img_data_height;

    /** Asignar espacio en la GPU para nuestra original img, new img, y dimensiones **/
    unsigned char *src_img, *out_img;
    hipMalloc( (void**)&src_img, img_data_size);
    hipMalloc( (void**)&out_img, img_data_size);

    
    /** Transferir memoria del host al device **/
    hipMemcpy(src_img, modified_img.data, img_data_size, hipMemcpyHostToDevice);
   
    /** Preparar argumentos dim3 para la GPU (threads per block & num of blocks)**/
    dim3 dimBlocks(N_THREADS, N_THREADS, 1); //hebras por bloque, bloque de 2 dimensiones
    dim3 dimGrid(ceil(img_data_width/N_THREADS), ceil(img_data_height/N_THREADS), 1); //numero de bloques (ceil para redondear valores al alza)

    auto start_time = chrono::system_clock::now(); //Tiempo inicio

    auto c = std::chrono::system_clock::now();

    /** Sobel mediante GPU **/
    filter_Sobel<<<dimGrid, dimBlocks>>>(src_img, out_img, img_data_width, img_data_height);

    chrono::duration<double> end_time = chrono::system_clock::now() - start_time; //Tiempo final
    printf("Execution time: \033[1;34m%*.5f\033[0m miliseconds\n",5,1000*end_time.count()); //Imprimir tiempo de ejecucion

    /** Copiar datos de vuelta al host **/
    hipMemcpy(modified_img.data, out_img, img_data_size, hipMemcpyDeviceToHost);

    /** Tamaño de nuestra imagen **/
    printf("\nProcessing %s: \033[1;34m%d\033[0m rows x \033[1;34m%d\033[0m columns\nTotal size: \033[1;34m%d\033[0m pixels \r\n", argv[1], img_data_width, img_data_height, img_data_size);

    /** Escribir imagen mediante OpenCV **/
    imwrite( "output_image.png", modified_img );

    /** Liberar memoria asignada previamente **/
    hipFree(src_img);
    hipFree(out_img);
    return 0;
}